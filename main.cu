#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

__global__
void matAddKernel(float** A, float** B, float** C, int n)
{
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	if(j<n)
	{ 
		for (int i = 0; i < n; ++i)
			A[i][j] = B[i][j] + C[i][j];
	}
}

__global__
void matAddKernel(float** A, float** B, float** C, int n)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if(i<n)
	{ 
		for (int j = 0; j < n; ++j)
			A[i][j] = B[i][j] + C[i][j];
	}
}

__global__
void matAddKernel(float** A, float** B, float** C, int n)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	if(i<n && j<n) 
		A[i][j] = B[i][j] + C[i][j];
}

void matAdd(float** A, float** B, float** C, int n)
{
	int size = n * n * sizeof(float);
	float **d_A, **d_B, **d_C;
	
	hipMalloc((void ***) &d_B, size);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
	hipMalloc((void ***) &d_C, size);
	hipMemcpy(d_C, C, size, hipMemcpyHostToDevice);
	
	hipMalloc((void ***) &d_A, size);
	
	matAddKernel<<<ceil(n/2560), 256>>>(d_A, d_B, d_C, n);
	
	hipMemcpy(A, d_A, size, hipMemcpyDeviceToHost);
	// Free device memory for A, B, C
	hipFree(d_A); hipFree(d_B); hipFree (d_C);
}









__global__
void matvecMultKernel(float* A, float** B, float* C, int n)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if(i<n) 
	{
		for (int j = 0; j < ; ++j)
			A[i] += B[i][j] + C[j];
	}
}

void matvecMult(float* A, float** B, float* C, int n)
{
	int size = n * sizeof(float);
	float *d_A, **d_B, *d_C;
	
	hipMalloc((void ***) &d_B, size*n);
	hipMemcpy(d_B, B, size*n, hipMemcpyHostToDevice);
	hipMalloc((void **) &d_C, size);
	hipMemcpy(d_C, C, size, hipMemcpyHostToDevice);
	
	hipMalloc((void ***) &d_A, size);
	
	matvecMultKernel<<<ceil(n/2560), 256>>>(d_A, d_B, d_C, n);
	
	hipMemcpy(A, d_A, size, hipMemcpyDeviceToHost);
	// Free device memory for A, B, C
	hipFree(d_A); hipFree(d_B); hipFree (d_C);
}


int main(int argc, char const *argv[])
{
	/* code */
	return 0;
}